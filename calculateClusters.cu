#include <stdio.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h> //powf to use in cuda device main thing

//__device__ int adding(int x, int y){
//    int result = x + y;
//    return result;
//}
//(devicePoints, deviceClusters, returnClusterNum, numDimensions, numClusters, numPoints);

//findClusters<<<1, numClusters>>>(devicePoints, deviceClusterAssignments, deviceClusters, numDimensions, numClusters, numPoints);
__global__ void calculateClusterCenter(float *devicePoints, int *deviceClusterAssignments, float *deviceClusters, int numDimensions, int numClusters, int numPoints){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int counter = 0;
    for(int i = 0; i < numPoints; i++){
        if(deviceClusterAssignments[i] == idx){
            for(int j = 0; j < numDimensions; j++){
                deviceClusters[idx * numDimensions + j] += devicePoints[i * numDimensions + j];
            }
            counter++;
        }
    }

    for(int i = 0; i < numDimensions; i++){
        printf("Cluster %d, Value: %f\n",idx, deviceClusters[idx * numDimensions + numDimensions]);
        deviceClusters[idx * numDimensions + numDimensions] = deviceClusters[idx * numDimensions + numDimensions] / counter;
    }



    /*
    float sum = 0.0, min_dist;
	int id_cluster_center = 0;

	for(int i = 0; i < numDimensions; i++){
		//sum += pow(clusters[0].getCentralValue(i) - point.getValue(i), 2.0);
        sum += powf(deviceClusters[i] - devicePoints[idx * numDimensions + i] , 2);
	}

	min_dist = sqrtf(sum);

	for(int i = 1; i < numClusters; i++){
		double dist;
		sum = 0.0;

		for(int j = 0; j < numDimensions; j++){
			sum += powf(deviceClusters[i * numDimensions + j] - devicePoints[idx * numDimensions + j] , 2);
		}

		dist = sqrtf(sum);

		if(dist < min_dist){
			min_dist = dist;
			id_cluster_center = i;
		}
	}
    //printf("%d",id_cluster_center);
	returnClusterNum[idx] = id_cluster_center;
    */
}

int main() {
    int numDimensions = 4;
    int numClusters = 3;
    int numPoints = 5;
    //float arrayPoints[];
    std::vector<std::vector<float>> points = {
        {8.8, 9.9, 0.0, 1.1},
        {2.1, 4.3, 6.5, 8.7},
        {4.4, 5.5, 6.6, 7.7},
        {4.4, 5.5, 6.6, 7.7},
        {8.8, 9.9, 0.0, 1.1}
    };
    std::vector<int> clusterAssignmentsIn = {1,1,2,1,0};
    std::vector<std::vector<float>> clusters = {
        {3.2, 3.4, 7.6, 2.8},
        {4.4, 5.5, 6.6, 7.7},
        {8.8, 9.9, 0.0, 1.1},
    };
    

    int pointsLengthNeeded = numDimensions * numPoints;
    int clustersLengthNeeded = numDimensions * numClusters;
    float pointsArray[pointsLengthNeeded];
    int clusterAssignmentsArray[numClusters];
    //float clustersArray[clustersLengthNeeded];
    float *devicePoints;
    int *deviceClusterAssignments;
    float *deviceClusters;
    
    for(int i = 0; i < numPoints; i++){
        for(int j = 0; j < numDimensions; j++){
            pointsArray[i * numDimensions + j] = points[i][j];
        }
        //changePoint++;
    }

    for(int i = 0; i < numClusters; i++){
        clusterAssignmentsArray[i] = clusterAssignmentsIn[i];//in the actual implementation just store everything in the array already so you dont need to copy it from vec to array plus the clusters calculations will just be in the same array you used
    }
   // for(int i = 0; i < numClusters; i++){
    //    for(int j = 0; j < numDimensions; j++){
    ///        clustersArray[i * numDimensions + j] = clusters[i][j];
    //    }
    //}



    //this stuff allocates the memory needed for the array
    hipMalloc((void**)&devicePoints, pointsLengthNeeded * sizeof(float));
    hipMalloc((void**)&deviceClusterAssignments, numClusters * sizeof(int));
    hipMalloc((void**)&deviceClusters, clustersLengthNeeded * sizeof(float));

    //yo basically this just copies the data from the host array to the device array to the position that is already allocated and it will be used
    hipMemcpy(devicePoints, pointsArray, pointsLengthNeeded * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceClusterAssignments, clusterAssignmentsArray, numClusters * sizeof(int), hipMemcpyHostToDevice);

    //this is one block of 10 threads - try multiple blocks and just get an understanding of it more than high level
    calculateClusterCenter<<<1, numClusters>>>(devicePoints, deviceClusterAssignments, deviceClusters, numDimensions, numClusters, numPoints);
    float clustersNewCenter[clustersLengthNeeded];
    hipMemcpy(clustersNewCenter, deviceClusters, clustersLengthNeeded * sizeof(float), hipMemcpyDeviceToHost);
    
    for(int i = 0; i < numClusters; i++){
        std::cout << "Cluster #" << i << ": ";
        for(int j = 0; j < numDimensions; j++){
            std::cout << clustersNewCenter[i * numDimensions + j] << " ";
        }
        std::cout << std::endl;
    }

    hipFree(devicePoints);
    hipFree(deviceClusterAssignments);
    hipFree(deviceClusters);
    
    return 0;
}